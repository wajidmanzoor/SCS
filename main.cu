#include "hip/hip_runtime.h"
#include <stdio.h>
#include "./src/Graph.h"
#include "./src/helpers.cc"


void cal_query_dist()
{

    // Description : Intialize querry distance array with INF
    q_dist = new ui[n];
    for(ui i =0;i<n;i++)
        q_dist[i] = INF;

    // Description: Queue that stores vertices
    queue<ui> Q;

    // Description : set distance of querry vertex as 0.
    q_dist[QID] = 0;

    // Description: Push querry vertex to Queue.
    Q.push(QID);

    // Description : Itterate till queue is empty
    while (!Q.empty()) {

        // Description : Get first vertex (v) from queue.
        ui v = Q.front();
        Q.pop();

        // Description: Iterate through the neighbors of V
        for(ui i = pstart[v]; i < pstart[v+1]; i++){
            ui w = edges[i];

            // Description : if distance of neighbor is INF, set to dstance of parent + 1.
            // Push neighbor to queue.
            if(q_dist[w] == INF){
                q_dist[w] = q_dist[v] + 1;
                Q.push(w);
            }
        }
    }
}



int main(int argc, const char * argv[] ) {

    if(argc!=6){
        cout<<"wrong input parameters!"<<endl;exit(1);
        exit(1);

    }
    // ./SCS ./graph.txt 6 9 2 100000
    N1 = atoi(argv[2]); //size LB
    N2 = atoi(argv[3]); //size UB
    QID = atoi(argv[4]); //Query vertex ID
    ui paritionSize = atoi(argv[5]);

    const char* filepath = argv[1];
    load_graph(filepath);

    
    Timer timer;
    StartTime = (double)clock() / CLOCKS_PER_SEC;

    core_decomposition_linear_list();

    // Description: upper bound defined
    ku = miv(core[QID], N2-1);
    kl = 0;
    ubD = N2-1;
    cal_query_dist();

    ui *deviceOffset,*deviceNeighbors,*deviceDegree, *deviceDistance,*deviceCore;
    ui *deviceLowerBoundDegree;

    /*cout << " d max " << dMAX<<endl;
    for(ui i=0;i<n;i++){
      if(core[i]==10){
        cout<<"Vertex "<<i<<" Core "<<core[i]<<endl;
      }
    }*/


    hipMalloc((void**)&deviceCore, n * sizeof(ui));
    hipMemcpy(deviceCore, core, n * sizeof(ui), hipMemcpyHostToDevice);

    hipMalloc((void**)&deviceDegree, n * sizeof(ui));
    hipMemcpy(deviceDegree, degree, n * sizeof(ui), hipMemcpyHostToDevice);

    hipMalloc((void**)&deviceOffset, (n+1) * sizeof(ui));
    hipMemcpy(deviceOffset, pstart, (n+1) * sizeof(ui), hipMemcpyHostToDevice);


    hipMalloc((void**)&deviceNeighbors, (2*m) * sizeof(ui));
    hipMemcpy(deviceNeighbors, edges, (2*m) * sizeof(ui), hipMemcpyHostToDevice);

     hipMalloc((void**)&deviceDistance, n * sizeof(ui));
    hipMemcpy(deviceDistance, q_dist, n * sizeof(ui), hipMemcpyHostToDevice);

    hipMalloc((void**)&deviceLowerBoundDegree, sizeof(ui));
    hipMemcpy(deviceLowerBoundDegree, &kl,sizeof(ui),hipMemcpyHostToDevice);

    ui *deviceIntialTaskList, *deviceIntialStatusList, *deviceGlobalCounter,*deviceEntries;

    ui INTOTAL_WARPS=32;
    ui intialParitionSize = (n/INTOTAL_WARPS)+1;
    ui intialSize = intialParitionSize*INTOTAL_WARPS;
    cout<<"Psize "<<intialParitionSize<<" Size "<<intialSize<<endl;

    hipMalloc((void**)&deviceIntialTaskList, intialSize*sizeof(ui));

    hipMalloc((void**)&deviceIntialStatusList, intialSize*sizeof(ui));

    hipMalloc((void**)&deviceGlobalCounter, sizeof(ui));
    hipMalloc((void**)&deviceEntries,INTOTAL_WARPS* sizeof(ui));

    ui globalCounter = 0;
    hipMemcpy(deviceGlobalCounter, &globalCounter, sizeof(ui), hipMemcpyHostToDevice);

    int shared_memory_size =  INTOTAL_WARPS* sizeof(ui);
    IntialReductionRules<<<1,BLK_DIM,shared_memory_size>>>(deviceOffset,deviceNeighbors,deviceDegree,deviceDistance,deviceCore,deviceIntialTaskList,deviceIntialStatusList,deviceEntries, deviceGlobalCounter,QID,n ,N2,kl,intialParitionSize);
    hipDeviceSynchronize();
    hipMemcpy(&globalCounter,deviceGlobalCounter,sizeof(ui),hipMemcpyDeviceToHost);
    cout<<" Total "<<globalCounter<<endl;
    /*ui *temp,*temp1,*temp2;
    temp = new ui[intialSize];
    temp1 = new ui[intialSize];
    temp2 = new ui[INTOTAL_WARPS];
    hipMemcpy(temp,deviceIntialTaskList,intialSize*sizeof(ui),hipMemcpyDeviceToHost);
    hipMemcpy(temp1,deviceIntialStatusList,intialSize*sizeof(ui),hipMemcpyDeviceToHost);
    hipMemcpy(temp2,deviceEntries,INTOTAL_WARPS*sizeof(ui),hipMemcpyDeviceToHost);
    ui s;
    for(int i = 0; i < INTOTAL_WARPS; i++){
      cout << "Entries "<<temp2[i]<<endl;
      s = intialParitionSize*i;
      for(int j=0;j<temp2[i];j++){
        cout << "Vertex "<<temp[s+j]<<" Satus "<<temp1[s+j]<<endl;
    }
    }*/


    ui *reducedTaskList, *reducedStatusList;

    hipMalloc((void**)&reducedTaskList, globalCounter*sizeof(ui));
    hipMalloc((void**)&reducedStatusList, globalCounter*sizeof(ui));

   

    CompressTask<<<1,BLK_DIM>>>(deviceIntialTaskList,deviceIntialStatusList,deviceEntries,reducedTaskList, reducedStatusList,intialParitionSize);
    hipDeviceSynchronize();


    hipFree(deviceIntialTaskList);
    hipFree(deviceIntialStatusList);
    hipFree(deviceEntries);
    hipFree(deviceGlobalCounter);

    /*ui *temp3,*temp4;
    temp3 = new ui[globalCounter];
    temp4 = new ui[globalCounter];

    hipMemcpy(temp3,reducedTaskList,globalCounter*sizeof(ui),hipMemcpyDeviceToHost);
    hipMemcpy(temp4,reducedStatusList,globalCounter*sizeof(ui),hipMemcpyDeviceToHost);

    cout<<"affter "<<endl;
    for(ui i =0;i<globalCounter;i++){
      cout<<"Vertex "<<temp3[i]<<"status " <<temp4[i]<<endl;
    }*/





    ui *taskOffset;

    taskOffset = new ui[paritionSize];
    memset(taskOffset, 0, paritionSize * sizeof(ui));
    taskOffset[1]= globalCounter;
    taskOffset[paritionSize-1] = 1;

    ui *deviceTaskList,*deviceStatusList, *deviceTaskOffset;

    hipMalloc((void**)&deviceTaskList, TOTAL_WARPS*paritionSize*sizeof(ui));
    hipMalloc((void**)&deviceStatusList, TOTAL_WARPS*paritionSize*sizeof(ui));

    hipMemcpy(deviceTaskList, reducedTaskList,globalCounter*sizeof(ui),hipMemcpyDeviceToDevice);
    hipMemcpy(deviceStatusList, reducedStatusList,globalCounter*sizeof(ui),hipMemcpyDeviceToDevice);

    hipMalloc((void**)&deviceTaskOffset, TOTAL_WARPS*paritionSize*sizeof(ui));
    hipMemcpy(deviceTaskOffset,taskOffset,paritionSize*sizeof(ui),hipMemcpyHostToDevice);

    hipFree(reducedTaskList);
    hipFree(reducedStatusList);

    bool *deviceStopFlag;
    bool stopFlag;

    hipMalloc((void**)&deviceStopFlag,sizeof(bool));

    shared_memory_size = WARPS_EACH_BLK * sizeof(ui);
    while(1){

        hipMemset(deviceStopFlag,1,sizeof(bool));
        hipMemcpy(&stopFlag,deviceStopFlag,sizeof(bool),hipMemcpyDeviceToHost);

        SCSSpeedEff <<<BLK_NUMS,BLK_DIM,shared_memory_size>>>(deviceTaskList, deviceStatusList,deviceTaskOffset,deviceNeighbors, deviceOffset, deviceDegree, deviceDistance, deviceStopFlag, deviceLowerBoundDegree, N1, N2, paritionSize, dMAX);

        hipMemcpy(&stopFlag,deviceStopFlag,sizeof(bool),hipMemcpyDeviceToHost);
        hipMemcpy(&kl, deviceLowerBoundDegree,sizeof(ui),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        if(stopFlag){
          cout<< " Max Min degree  "<<kl<<endl;
          cout<<"time = "<<integer_to_string(timer.elapsed()).c_str()<<endl;
            break;
        }
      }
    hipFree(deviceOffset);
    hipFree(deviceNeighbors);
    hipFree(deviceDegree);
    hipFree(deviceDistance);
    hipFree(deviceCore);

    hipFree(deviceLowerBoundDegree);
    hipFree(deviceTaskList);
    hipFree(deviceStatusList);
    hipFree(deviceTaskOffset);
    hipFree(deviceStopFlag);


    return 0;

}