#include "hip/hip_runtime.h"
#include "./inc/heuristic.h"
#include "./src/gpuMemoryAllocation.cc"
#include "./src/helpers.cc"
#include <unistd.h>  



#define CUDA_CHECK_ERROR(kernelName) { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("CUDA Error in kernel %s, file %s at line %d: %s\n", kernelName, __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

struct subtract_functor {
  const ui x;

  subtract_functor(ui _x): x(_x) {}

  __host__ __device__ ui operator()(const ui & y) const {
    return y - x;
  }
};

bool isServerExit(const string & str) {
  string trimmedStr = str;
  trimmedStr.erase(trimmedStr.find_last_not_of(" \n\r\t") + 1);
  trimmedStr.erase(0, trimmedStr.find_first_not_of(" \n\r\t"));

  transform(trimmedStr.begin(), trimmedStr.end(), trimmedStr.begin(), ::tolower);

  return trimmedStr == "server_exit";
}

void listenForMessages() {

  msg_queue_server server('g');
  long type = 1;
  //cout<<"rank "<<worldRank<<" listen here "<<endl;
  ui id =0;
  while (true) {
    if (server.recv_msg(type)) {

      string msg = server.get_msg();
      queryInfo query(id, msg);
      //totalQuerry++;
      id++;
      messageQueueMutex.lock();
      messageQueue.push_back(query);
      messageQueueMutex.unlock();
      if (isServerExit(msg))
        break;
    }
  }
}

inline void preprocessQuery(string msg, ui queryId) {
  istringstream iss(msg);
  vector < ui > argValues;
  ui number, countArgs;
  countArgs = 0;
  while (iss >> number) {
    argValues.push_back(number);
    countArgs++;
  }
  int ind = -1;
  for (int x = 0; x < limitQueries; x++) {
    if (queries[x].solFlag != 0) {
      ind = x;
      break;
    }
  }

  //cout<<"Rank: "<<worldRank<<" Ind "<<ind<<" msg "<<msg<<endl;

  queries[ind].updateQueryData(argValues[0], argValues[1], argValues[2], argValues[3], argValues[4],queryId, ind);
  if (queries[ind].isHeu)
    CSSC_heu(ind);
  //cout <<"Rank "<<worldRank<< " : Processing : " << queries[ind] << endl;
  if (queries[ind].kl == queries[ind].ku) {
    cout <<"Rank "<<worldRank<< " : heuristic find the OPT!" << endl;
    cout <<"Rank "<<worldRank<< " : Found Solution : " << queries[ind] << endl;
    stringstream ss;
    ss <<queries[ind].N1<< "|" << queries[ind].N2 << "|"<< queries[ind].QID << "|"<< integer_to_string(queries[ind].receiveTimer.elapsed()).c_str() << "|"<< queries[ind].kl << "|"<<"0"<< "|"<<"1\n";
    string output = ss.str();
    MPI_File_write(fh, output.c_str(), output.length(), MPI_BYTE, &fstatus);
    queries[ind].solFlag = 1;
    numQueriesProcessing--;
    
  }else{
    cal_query_dist(queries[ind].QID);
    chkerr(hipMemcpy(deviceGraph.degree + (ind * n), degree, n * sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(deviceGraph.distance + (ind * n), q_dist, n * sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(deviceGraph.lowerBoundDegree + ind, & (queries[ind].kl), sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(deviceGraph.lowerBoundSize + ind, & (queries[ind].N1), sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(deviceGraph.upperBoundSize + ind, & (queries[ind].N2), sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(deviceGraph.limitDoms + ind, & (queries[ind].limitDoms), sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMemset(initialTask.globalCounter, 0, sizeof(ui)));
    chkerr(hipMemset(initialTask.entries, 0, INTOTAL_WARPS * sizeof(ui)));
    chkerr(hipMemset(deviceGraph.newOffset + ((n + 1) * ind), 0, (n + 1) * sizeof(ui)));

    if (queries[ind].kl <= 1)
      queries[ind].ubD = queries[ind].N2 - 1;
    else {
      for (ui d = 1; d <= queries[ind].N2; d++) {
        if (d == 1 || d == 2) {
          if (queries[ind].kl + d > queries[ind].N2) {
            queries[ind].ubD = d - 1;
            break;
          }
        } else {
          ui min_n = queries[ind].kl + d + 1 + floor(d / 3) * (queries[ind].kl - 2);
          if (queries[ind].N2 < min_n) {
            queries[ind].ubD = d - 1;
            break;
          }
        }
      }
    }
    maxN2 = mav(maxN2, queries[ind].N2);

    //queries[ind].receiveTimer.restart();

    initialReductionRules << < BLK_NUM2, BLK_DIM2, sharedMemorySizeinitial >>> (deviceGenGraph, deviceGraph, initialTask, n, queries[ind].ubD, initialPartitionSize, ind);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Intial Reduction ");

    ui globalCounter;
    chkerr(hipMemcpy( & globalCounter, initialTask.globalCounter, sizeof(ui), hipMemcpyDeviceToHost));

    ui writeWarp, ntasks, space;
    chkerr(hipMemcpy( &writeWarp, deviceTask.sortedIndex, sizeof(ui), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy( &ntasks, deviceTask.numTasks + writeWarp, sizeof(ui), hipMemcpyDeviceToHost));
    ui offsetPsize = partitionSize/factor;
    chkerr(hipMemcpy( &space, deviceTask.taskOffset + (writeWarp*offsetPsize + ntasks) , sizeof(ui), hipMemcpyDeviceToHost));
    if(globalCounter>=(partitionSize-space)){
      //cout <<"Rank "<<worldRank<<" : "<< "Intial Task > partition Size " << msg << endl;
      

    }else{
      CompressTask << < BLK_NUM2, BLK_DIM2 >>> (deviceGenGraph, deviceGraph, initialTask, deviceTask, initialPartitionSize, queries[ind].QID, ind, n, partitionSize, TOTAL_WARPS, factor);
      hipDeviceSynchronize();
      CUDA_CHECK_ERROR("Compress ");

      thrust::inclusive_scan(thrust::device_ptr < ui > (deviceGraph.newOffset + ((n + 1) * ind)), thrust::device_ptr < ui > (deviceGraph.newOffset + ((n + 1) * (ind + 1))), thrust::device_ptr < ui > (deviceGraph.newOffset + ((n + 1) * ind)));
      hipDeviceSynchronize();


      NeighborUpdate << < BLK_NUMS, BLK_DIM , sharedMemoryUpdateNeigh >>> (deviceGenGraph, deviceGraph,deviceTask, TOTAL_WARPS, ind, n, m,partitionSize,factor);
      hipDeviceSynchronize();
      CUDA_CHECK_ERROR("Neighbor  ");
      thrust::device_ptr<ui> d_sortedIndex_ptr(deviceTask.sortedIndex);
      thrust::device_ptr<ui> d_mapping_ptr(deviceTask.mapping);

      thrust::device_vector<ui> d_temp_input(d_sortedIndex_ptr, d_sortedIndex_ptr + TOTAL_WARPS);

      thrust::sequence(thrust::device, d_sortedIndex_ptr, d_sortedIndex_ptr + TOTAL_WARPS);

      thrust::sort_by_key(thrust::device,
                          d_temp_input.begin(), d_temp_input.end(),
                          d_sortedIndex_ptr,
                          thrust::less<ui>());

      thrust::scatter(thrust::device,
                      thrust::make_counting_iterator<ui>(0),
                      thrust::make_counting_iterator<ui>(TOTAL_WARPS),
                      d_sortedIndex_ptr,
                      d_mapping_ptr);


      thrust::transform(thrust::device, d_mapping_ptr, d_mapping_ptr + TOTAL_WARPS, d_mapping_ptr, subtract_from(TOTAL_WARPS-1));
    }
    
  }
  
}

inline void processQueries() {
  chkerr(hipMemset(deviceGraph.flag, 0, limitQueries * sizeof(ui)));
  chkerr(hipMemset(deviceTask.doms, 0, TOTAL_WARPS * partitionSize * sizeof(ui)));
  
  sharedMemorySizeTask = 2 * WARPS_EACH_BLK * sizeof(ui) + WARPS_EACH_BLK * sizeof(int) + WARPS_EACH_BLK * sizeof(double) + 2 * WARPS_EACH_BLK * sizeof(ui) + maxN2 * WARPS_EACH_BLK * sizeof(ui);

  ProcessTask << < BLK_NUMS, BLK_DIM, sharedMemorySizeTask >>> (
        deviceGenGraph, deviceGraph, deviceTask, partitionSize, factor,maxN2, n, m,dMAX,limitQueries, red1, red2, red3 , prun1, prun2);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("Process Task");
  chkerr(hipMemset(deviceTask.doms, 0, TOTAL_WARPS * partitionSize * sizeof(ui)));
  // This kernel identifies vertices dominated by ustar and sorts them in decreasing order of their connection score.
  FindDoms << < BLK_NUMS, BLK_DIM, sharedMemorySizeDoms >>> (
    deviceGenGraph, deviceGraph, deviceTask, partitionSize,factor, n, m,dMAX,limitQueries);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("Find Doms");

  // This kernel writes new tasks, based on ustar and the dominating set, into the task array or buffer. It reads from the buffer and writes to the task array.
  //ui flag = 1 ;
  //chkerr(hipMemcpy( &flag, &deviceBuffer.outOfMemoryFlag, sizeof(ui),hipMemcpyDeviceToHost));

  Expand << < BLK_NUMS, BLK_DIM, sharedMemorySizeExpand >>> (
        deviceGenGraph, deviceGraph, deviceTask, deviceBuffer, partitionSize,factor, copyLimit, bufferSize, numTaskHost-numReadHost, readLimit, n, m, dMAX,limitQueries);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("Expand ");
  RemoveCompletedTasks<<<BLK_NUMS, BLK_DIM>>>( deviceGraph,deviceTask, partitionSize,factor);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("Remove Completed ");


  chkerr(hipMemcpy(&(outMemFlag), deviceBuffer.outOfMemoryFlag, sizeof(ui),hipMemcpyDeviceToHost));
    
  chkerr(hipMemcpy( & tempHost, deviceBuffer.temp, sizeof(ui),
    hipMemcpyDeviceToHost));
  chkerr(hipMemcpy( & numReadHost, deviceBuffer.numReadTasks, sizeof(ui),
    hipMemcpyDeviceToHost));
  chkerr(hipMemcpy( & numTaskHost, deviceBuffer.numTask, sizeof(ui),
    hipMemcpyDeviceToHost));

  chkerr(hipMemcpy(queryStopFlag, deviceGraph.flag, limitQueries * sizeof(ui), hipMemcpyDeviceToHost));

  for (ui i = 0; i < limitQueries; i++) {
    if ((queryStopFlag[i] == 0) && (queries[i].solFlag == 0)) {
      chkerr(hipMemcpy( & (queries[i].numRead), deviceGraph.numRead + i, sizeof(ui), hipMemcpyDeviceToHost));
      chkerr(hipMemcpy( & (queries[i].numWrite), deviceGraph.numWrite + i, sizeof(ui), hipMemcpyDeviceToHost));
      if ((queries[i].numRead == queries[i].numWrite)) {
        chkerr(hipMemcpy( & (queries[i].kl), deviceGraph.lowerBoundDegree + i, sizeof(ui), hipMemcpyDeviceToHost));
        cout <<"Rank "<<worldRank<<" : Found Solution : " << queries[i] << endl;
        stringstream ss;
        ss <<queries[i].N1<< "|" << queries[i].N2 << "|"<< queries[i].QID << "|"<< integer_to_string(queries[i].receiveTimer.elapsed()).c_str() << "|"<< queries[i].kl << "|"<<"0"<< "|"<<"0\n";
        string output = ss.str();
        MPI_File_write(fh, output.c_str(), output.length(), MPI_BYTE, &fstatus);
        //Send result Data to Rank 0 system 
        queries[i].solFlag = 1;
        numQueriesProcessing--;
      }
    }

  }
   if(outMemFlag){
      for (ui i = 0; i < limitQueries; i++) {
        if ( queries[i].solFlag==0) {
        chkerr(hipMemcpy( & (queries[i].kl), deviceGraph.lowerBoundDegree + i, sizeof(ui), hipMemcpyDeviceToHost));
        cout <<"Rank "<<worldRank<<" : Buffer out of memory ! Found Solution : " << queries[i] << endl;
        stringstream ss;
        ss <<queries[i].N1<< "|" << queries[i].N2 << "|"<< queries[i].QID << "|"<< integer_to_string(queries[i].receiveTimer.elapsed()).c_str() << "|"<< queries[i].kl << "|"<<"1"<< "|"<<"0\n";

        string output = ss.str();
        MPI_File_write(fh, output.c_str(), output.length(), MPI_BYTE, &fstatus);
        queries[i].solFlag = 1;
        numQueriesProcessing--;
          
        }
      }
    }

  if (numTaskHost == numReadHost) {
    chkerr(hipMemset(deviceBuffer.numTask, 0, sizeof(ui)));
    chkerr(hipMemset(deviceBuffer.numReadTasks, 0, sizeof(ui)));
    chkerr(hipMemset(deviceBuffer.temp, 0, sizeof(ui)));
    chkerr(hipMemset(deviceBuffer.writeMutex, 0, sizeof(ui)));
    chkerr(hipMemset(deviceBuffer.readMutex, 0, sizeof(ui)));
    chkerr(hipMemset(deviceBuffer.taskOffset, 0, (numReadHost + 1) * sizeof(ui)));
  }

  // If the number of tasks written to the buffer exceeds the number read at this level, left shift the tasks that were written but not read to the start of the array.
  if ((numReadHost < numTaskHost) && (numReadHost > 0)) {
    cout << "Num read " << numReadHost << " num Task " << numTaskHost << endl;
    chkerr(hipMemcpy( & startOffset, deviceBuffer.taskOffset + numReadHost,
      sizeof(ui), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy( & endOffset, deviceBuffer.taskOffset + numTaskHost, sizeof(ui),
      hipMemcpyDeviceToHost));

    thrust::transform(
      thrust::device_ptr < ui > (deviceBuffer.taskOffset + numReadHost),
      thrust::device_ptr < ui > (deviceBuffer.taskOffset + numTaskHost + 1),
      thrust::device_ptr < ui > (deviceBuffer.taskOffset),
      subtract_functor(startOffset));

    chkerr(hipMemset(deviceBuffer.taskOffset + (numTaskHost - numReadHost + 1), 0,
      numReadHost * sizeof(ui)));

    thrust::copy(thrust::device_ptr < ui > (deviceBuffer.size + numReadHost),
      thrust::device_ptr < ui > (deviceBuffer.size + numTaskHost),
      thrust::device_ptr < ui > (deviceBuffer.size));

    thrust::copy(thrust::device_ptr < ui > (deviceBuffer.queryIndicator + numReadHost),
      thrust::device_ptr < ui > (deviceBuffer.queryIndicator + numTaskHost),
      thrust::device_ptr < ui > (deviceBuffer.queryIndicator));

    thrust::copy(
      thrust::device_ptr < ui > (deviceBuffer.taskList + startOffset),
      thrust::device_ptr < ui > (deviceBuffer.taskList + endOffset),
      thrust::device_ptr < ui > (deviceBuffer.taskList));

    thrust::copy(
      thrust::device_ptr < ui > (deviceBuffer.statusList + startOffset),
      thrust::device_ptr < ui > (deviceBuffer.statusList + endOffset),
      thrust::device_ptr < ui > (deviceBuffer.statusList));

    int justCheck = (int)(numTaskHost - numReadHost);

    chkerr(hipMemcpy(deviceBuffer.numTask, & justCheck, sizeof(ui),
      hipMemcpyHostToDevice));
    chkerr(hipMemcpy(deviceBuffer.temp, & justCheck, sizeof(ui),
      hipMemcpyHostToDevice));
    chkerr(hipMemset(deviceBuffer.writeMutex, 0, sizeof(ui)));
    chkerr(hipMemset(deviceBuffer.numReadTasks, 0, sizeof(ui)));

    chkerr(hipMemset(deviceBuffer.readMutex, 0, sizeof(ui)));
  }
  chkerr(hipMemset(deviceTask.doms, 0, TOTAL_WARPS * partitionSize * sizeof(ui)));

   thrust::device_ptr<ui> d_sortedIndex_ptr(deviceTask.sortedIndex);
    thrust::device_ptr<ui> d_mapping_ptr(deviceTask.mapping);

    thrust::device_vector<ui> d_temp_input(d_sortedIndex_ptr, d_sortedIndex_ptr + TOTAL_WARPS);

    thrust::sequence(thrust::device, d_sortedIndex_ptr, d_sortedIndex_ptr + TOTAL_WARPS);

    thrust::sort_by_key(thrust::device,
                        d_temp_input.begin(), d_temp_input.end(),
                        d_sortedIndex_ptr,
                        thrust::less<ui>());

    thrust::scatter(thrust::device,
                    thrust::make_counting_iterator<ui>(0),
                    thrust::make_counting_iterator<ui>(TOTAL_WARPS),
                    d_sortedIndex_ptr,
                    d_mapping_ptr);


    thrust::transform(thrust::device, d_mapping_ptr, d_mapping_ptr + TOTAL_WARPS, d_mapping_ptr, subtract_from(TOTAL_WARPS-1));


}




void processMessageMasterServer() {
  bool stopListening = false;

  vector <MPI_Request> requests(worldSize);
  vector <MPI_Status> status(worldSize);
  
  vector < SystemInfo > systems(worldSize);
  vector < int > nQP(worldSize);

  vector <MPI_Request> endRequests(worldSize);
  vector <MPI_Status> endStatus(worldSize);
  vector <int> endFlag(worldSize);
  vector <SystemStatus> systemStatus(worldSize);
  for(int i = 0; i <worldSize;i++){
    endFlag[i] = 0;
    systemStatus[i] = IDLE;
  }

  systems[0] = {0, 0, 0};
  nQP[0] = 0;
  for (int i = 1; i < worldSize; i++) {
    systems[i] = { i, 0, 1 };
    nQP[i] = 0;
  }
  ui id =0;
  //cout<<"rank "<<worldRank<<" process here "<<endl;
  while (true) {

      systems[0].numQueriesProcessing = numQueriesProcessing;

      for (int i = 1; i < worldSize; i++) {
        
        if (systems[i].flag) {

          MPI_Irecv( & nQP[i], 1, MPI_INT, i, TAG_NQP, MPI_COMM_WORLD, & requests[i]);
        }

        MPI_Test( &requests[i], & systems[i].flag, & status[i]);
        if (systems[i].flag) {
          systems[i].numQueriesProcessing = nQP[i];
          //cout<<"Rank "<<worldRank<<" : Num Processing of system "<<i<<" updated to "<<systems[i].numQueriesProcessing<<endl;

        }

    }
    auto leastLoadedSystem = *std::min_element(systems.begin(), systems.end(),
      [](const SystemInfo & a,
        const SystemInfo & b) {
        return a.numQueriesProcessing < b.numQueriesProcessing;
      });
    leastQuery = leastLoadedSystem.numQueriesProcessing;

    

    

    if (!stopListening) {
      messageQueueMutex.lock();
      while ((!messageQueue.empty()) && (leastQuery < limitQueries)) {

        //cout<<"Rank with : "<<leastLoadedSystem.rank<<" Least "<<leastQuery<<" limit "<<limitQueries<<endl;
        
        queryInfo message = messageQueue.front();

        messageQueue.erase(messageQueue.begin());
        messageQueueMutex.unlock();
        ui queryId = message.queryId;
        string msg = message.queryString;
        if (isServerExit(msg)) {
          stopListening = true;
          for (int i = 1; i < worldSize; i++) {
            MessageType msgType = TERMINATE;
            MPI_Send( & msgType, 1, MPI_INT, i, TAG_MTYPE, MPI_COMM_WORLD);
          }
        } else {
          systems[0].numQueriesProcessing = numQueriesProcessing;
          nQP[0] = numQueriesProcessing;
          for (int i = 1; i < worldSize; i++) {
        
            if (systems[i].flag) {

              MPI_Irecv( & nQP[i], 1, MPI_INT, i, TAG_NQP, MPI_COMM_WORLD, & requests[i]);
            }

            MPI_Test( &requests[i], & systems[i].flag, & status[i]);
            if (systems[i].flag) {
              systems[i].numQueriesProcessing = nQP[i];

            }


          }

          auto leastLoadedSystem = *std::min_element(systems.begin(), systems.end(),
            [](const SystemInfo & a,
              const SystemInfo & b) {
              return a.numQueriesProcessing < b.numQueriesProcessing;
            });
          //cout<<"Rank "<<worldRank<<" : System with min np "<<leastLoadedSystem.rank<<endl;
          if (leastLoadedSystem.rank == 0) {
            //cout<<"Rank 0 : Processed itself.  msg :  "<<msg<<endl;
            numQueriesProcessing++;

            preprocessQuery(msg,id);
            id++;

          } else {

            if(systemStatus[leastLoadedSystem.rank] == IDLE){
              systemStatus[leastLoadedSystem.rank] = PROCESSING;
              endFlag[leastLoadedSystem.rank] = 1;

            }
            //msg.erase(std::remove(msg.begin(), msg.end(), '\n'), msg.end());

            //cout<<"Rank 0 : Sending to rank "<<leastLoadedSystem.rank<<" msg "<<msg<<endl;
            MessageType msgType = PROCESS_MESSAGE;
            MPI_Send( & msgType, 1, MPI_INT, leastLoadedSystem.rank, TAG_MTYPE, MPI_COMM_WORLD);
            MPI_Send(msg.c_str(), msg.length(), MPI_CHAR, leastLoadedSystem.rank, TAG_MSG, MPI_COMM_WORLD);
            systems[leastLoadedSystem.rank].numQueriesProcessing++;

            // Get confirmation 

          }

        }

        systems[0].numQueriesProcessing = numQueriesProcessing;

      for (int i = 1; i < worldSize; i++) {
        
        if (systems[i].flag) {

          MPI_Irecv( & nQP[i], 1, MPI_INT, i, TAG_NQP, MPI_COMM_WORLD, & requests[i]);
        }

        MPI_Test( &requests[i], & systems[i].flag, & status[i]);
        if (systems[i].flag) {
          systems[i].numQueriesProcessing = nQP[i];
          //cout<<"Rank "<<worldRank<<" : Num Processing of system "<<i<<" updated to "<<systems[i].numQueriesProcessing<<endl;

        }

    }
    auto leastLoadedSystem = *std::min_element(systems.begin(), systems.end(),
      [](const SystemInfo & a,
        const SystemInfo & b) {
        return a.numQueriesProcessing < b.numQueriesProcessing;
      });
    leastQuery = leastLoadedSystem.numQueriesProcessing;


        messageQueueMutex.lock();
      }
      messageQueueMutex.unlock();
    }

    if (numQueriesProcessing != 0) {
      processQueries();
    }
    
    for(int i =1 ; i < worldSize ; i ++){
      if(systemStatus[i]==PROCESSING){
        if (endFlag[i]) {

              MPI_Irecv( &systemStatus[i], 1, MPI_INT, i,TAG_TERMINATE, MPI_COMM_WORLD, & endRequests[i]);
              //cout<<"Rank "<<worldRank<<" : Recieved terminate from system "<<i<<endl;
          }

        MPI_Test( &endRequests[i], &endFlag[i], & endStatus[i]);


      }
       
    }

    if ((numQueriesProcessing == 0) && (stopListening)){
      bool allTerminatedOrIdle = std::all_of(systemStatus.begin(), systemStatus.end(), [](SystemStatus status) { return status == SystemStatus::TERMINATED || status == SystemStatus::IDLE; });
      if (allTerminatedOrIdle)
        break;
      
    }



  }

}

void processMessageOtherServer() {
  int flag = true;
  MPI_Request request;
  MPI_Status status;
  bool stopListening = false;
  MessageType msgType;
  int old = 0;
  ui id = 0;
  while (true) {
    if ((!stopListening) && (numQueriesProcessing < limitQueries)){
      if (flag) {
        MPI_Irecv( & msgType, 1, MPI_INT, 0, TAG_MTYPE, MPI_COMM_WORLD, & request);
      }

      MPI_Test( & request, & flag, & status);
      if (flag) {
        if (msgType == TERMINATE) {
          stopListening = true;

        } else {
          char buffer[1024];
          MPI_Recv(buffer, 1024, MPI_CHAR, 0, TAG_MSG, MPI_COMM_WORLD, & status);

          int count;
          MPI_Get_count(&status, MPI_CHAR, &count);
          buffer[count] = '\0';
          string msg(buffer);

          //cout<<"Rank "<<worldRank<<" : Recieved from  rank 0  msg "<<msg<<endl;
          numQueriesProcessing++;

          if(old != numQueriesProcessing){
            MPI_Send( &numQueriesProcessing, 1, MPI_INT, 0, TAG_NQP, MPI_COMM_WORLD);
            old = numQueriesProcessing;
            //cout<<"Rank "<<worldRank<<" : Num Processing updated to  "<<numQueriesProcessing<<endl;
          }
          preprocessQuery(msg,id);
          id ++;
          if(old != numQueriesProcessing){
            MPI_Send( &numQueriesProcessing, 1, MPI_INT, 0, TAG_NQP, MPI_COMM_WORLD);
            old = numQueriesProcessing;
            //cout<<"Rank "<<worldRank<<" : Num Processing updated to  "<<numQueriesProcessing<<endl;
          }
        }
      }
    }

    if (numQueriesProcessing != 0) {
      processQueries();

    }

    if ((numQueriesProcessing == 0) && (stopListening))
    {
      SystemStatus ss = TERMINATED;
      MPI_Send( &ss, 1, MPI_INT, 0 , TAG_TERMINATE, MPI_COMM_WORLD);
      //cout<<"Rank "<<worldRank<<" : Send terminate "<<endl;
      break;

    }

  }
}


int main(int argc,const char * argv[]) {
  if (argc != 13) {
    cerr << "Server wrong input parameters!" << endl;
    exit(1);
  }

  char** new_argv = new char*[argc];
  for (int i = 0; i < argc; i++) {
      new_argv[i] = const_cast<char*>(argv[i]);
  }

  int mpi_init_result = MPI_Init( & argc, & new_argv);
  if (mpi_init_result != MPI_SUCCESS) {
    cerr << "Error initializing MPI." << endl;
    return 1;
  }

  MPI_Comm_size(MPI_COMM_WORLD, & worldSize);
  MPI_Comm_rank(MPI_COMM_WORLD, & worldRank);
  graphPath = argv[1];
  size_t pos = graphPath.find_last_of("/\\");
  fileName = (pos != string::npos) ? graphPath.substr(pos + 1) : graphPath;

  fileName = "./results/exp9/" + fileName+"/"+to_string(worldSize)+".txt";

  MPI_File_open(MPI_COMM_WORLD, fileName, MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fh);
  if(worldRank==0){
    string header = "N1|N2|QID|Time|Degree|Overtime|Heu\n";
    MPI_File_write(fh, header.c_str(), header.length(), MPI_BYTE, &fstatus);
  }
  
  cout<<"rank "<<worldRank<<" Size "<<worldSize<<endl;

  const char * filepath = argv[1]; // Path to the graph file. The graph should be represented as an edge list with tab (\t) separators
  partitionSize = atoi(argv[2]); // Defines the partition size, in number of elements, that a single warp will read from and write to.
  bufferSize = atoi(argv[3]); // Specifies the size, in number of elements, where warps will write in case the partition overflows.
  copyLimit = stod(argv[4]); // Specifies that only warps with at most this percentage of their partition space filled will read from the buffer and write to their partition.
  readLimit = atoi(argv[5]); // Maximum number of tasks a warp with an empty partition can read from the buffer.
  limitQueries = atoi(argv[6]);
  factor = atoi(argv[7]);
  red1 = atoi(argv[8]);
  red2 = atoi(argv[9]);
  red3 = atoi(argv[10]);
  prun1 = atoi(argv[11]);
  prun2 = atoi(argv[12]);

  queries = new queryData[limitQueries];

  for (ui i = 0; i < limitQueries; i++) {
    queryData query;
    queries[i] = query;
  }

  load_graph(filepath);
  core_decomposition_linear_list();

  memoryAllocationGenGraph(deviceGenGraph);
  memoryAllocationGraph(deviceGraph, limitQueries);

  totalQuerry = 0;
  q_dist = new ui[n];
  outMemFlag = 0;

  maxN2 = 0;
  
  if (n <= WARPSIZE) {
        BLK_DIM2 = WARPSIZE;
        BLK_NUM2 = 1;
  } else if (n <= BLK_NUMS) {
      BLK_DIM2 = std::ceil(static_cast<float>(n) / WARPSIZE) * WARPSIZE;
      BLK_NUM2 = 1;
  } else {
      BLK_DIM2 = BLK_DIM;
      BLK_NUM2 = std::min(BLK_NUMS, static_cast<int>(std::ceil(static_cast<float>(n) / BLK_DIM2)));
  }

  INTOTAL_WARPS = (BLK_NUM2 * BLK_DIM2) / WARPSIZE;

  initialPartitionSize = static_cast<ui>(std::ceil(static_cast<float>(n) / INTOTAL_WARPS));

  memoryAllocationinitialTask(initialTask, INTOTAL_WARPS, initialPartitionSize);
  memoryAllocationTask(deviceTask, TOTAL_WARPS, partitionSize, limitQueries, factor);
  memoryAllocationBuffer(deviceBuffer, bufferSize, limitQueries, factor);

  sharedMemorySizeinitial = INTOTAL_WARPS * sizeof(ui);
  sharedMemoryUpdateNeigh = WARPS_EACH_BLK * sizeof(ui);

  queryStopFlag = new ui[limitQueries];

  memset(queryStopFlag, 0, limitQueries * sizeof(ui));

  sharedMemorySizeDoms = WARPS_EACH_BLK * sizeof(ui);
  sharedMemorySizeExpand = WARPS_EACH_BLK * sizeof(ui);

  numTaskHost = 0;
  numReadHost = 0;
  tempHost = 0;
  startOffset = 0;
  endOffset = 0;
  numQueriesProcessing = 0;
  
  if (worldRank == 0) {
    leastQuery = 0;
    thread listener(listenForMessages);
    thread processor(processMessageMasterServer);
    listener.join();
    processor.join();
    MPI_Finalize();
  } else {
    processMessageOtherServer();
    MPI_Finalize();
  }

  MPI_File_close(&fh);
  hipDeviceSynchronize();
  freeGenGraph(deviceGenGraph);
  freeGraph(deviceGraph);
  freeInterPointer(initialTask);
  freeTaskPointer(deviceTask);
  freeBufferPointer(deviceBuffer);

  return 0;
}
