#include "hip/hip_runtime.h"
#include "./inc/heuristic.h"
#include "./src/gpuMemoryAllocation.cc"
#include "./src/helpers.cc"

#define CUDA_CHECK_ERROR(kernelName) { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("CUDA Error in kernel %s, file %s at line %d: %s\n", kernelName, __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

int c =0;
bool serverQuit = false;

bool fileExists(const string& filename) {
    struct stat buffer;
    return (stat(filename.c_str(), &buffer) == 0);
}

// Function to write or append data to the file
void writeOrAppend(const string& filename, const string& data) {
    ofstream file;
    
    // Check if the file exists
    if (fileExists(filename)) {
        // Open the file in append mode if it exists
        file.open(filename, ios::app);
    } else {
        // Open the file in write mode if it doesn't exist
        file.open(filename);
    }
    
    if (file.is_open()) {
        file << data << endl;
        file.close();
    } else {
        cerr << "Unable to open the file." << endl;
    }
}
struct subtract_functor {
  const ui x;

  subtract_functor(ui _x): x(_x) {}

  __host__ __device__ ui operator()(const ui & y) const {
    return y - x;
  }
};

bool isServerExit(const std::string& str) {
    std::string trimmedStr = str;
    trimmedStr.erase(trimmedStr.find_last_not_of(" \n\r\t") + 1);
    trimmedStr.erase(0, trimmedStr.find_first_not_of(" \n\r\t"));

    std::transform(trimmedStr.begin(), trimmedStr.end(), trimmedStr.begin(), ::tolower);

    return trimmedStr == "server_exit";
}



void listenForMessages() {

  msg_queue_server server('g');
  long type = 1;
  while (true) {
    if (server.recv_msg(type)) {
      string msg = server.get_msg();
      queryInfo query(totalQuerry, msg);
      totalQuerry++;
      messageQueueMutex.lock();
      messageQueue.push_back(query);
      messageQueueMutex.unlock();
      if(isServerExit(msg))
      break;
    }
  }
}

void processMessages() {
  while (true) {
    messageQueueMutex.lock();
    while ( (!messageQueue.empty()) && (numQueriesProcessing < limitQueries)) {
      queryInfo message = messageQueue.front();
      messageQueue.erase(messageQueue.begin());
      messageQueueMutex.unlock();

        ui queryId = message.queryId;
        string queryText = message.queryString;

        istringstream iss(queryText);
        vector<ui> argValues;
        ui number, countArgs;
        countArgs = 0;
        if(isServerExit(queryText)){
          serverQuit = true;
          break;
        }

        while (iss >> number) {
          argValues.push_back(number);
          countArgs++;
        }

        if (countArgs != 5) {
          cout << "Client wrong input parameters! " << message << endl;
          continue;
        }
        int ind  = -1;
        for(ui x =0; x < limitQueries;x++){
          if(queries[x].solFlag!=0){
            ind = x;
            break;
          }
        }

        queries[ind].updateQueryData(argValues[0], argValues[1], argValues[2], argValues[3], argValues[4],queryId,ind);
        if (queries[ind].isHeu)
          CSSC_heu(ind);
        if (queries[ind].kl == queries[ind].ku) {
          stringstream ss;
          ss <<queries[ind].N1<< "|" << queries[ind].N2 << "|"<< queries[ind].QID << "|"<< integer_to_string(queries[ind].receiveTimer.elapsed()).c_str() << "|"<< queries[ind].kl << "|"<<"0"<< "|"<<"1";
          writeOrAppend(fileName,ss.str());
          cout << "heuristic find the OPT!" << endl;
          cout << "Found Solution : " << queries[ind] << endl;
          queries[ind].solFlag = 1;
          continue;
        }

        cal_query_dist(queries[ind].QID);
        chkerr(hipMemcpy(deviceGraph.degree + (ind * n), degree, n * sizeof(ui), hipMemcpyHostToDevice));
        chkerr(hipMemcpy(deviceGraph.distance + (ind * n), q_dist, n * sizeof(ui), hipMemcpyHostToDevice));
        chkerr(hipMemcpy(deviceGraph.lowerBoundDegree + ind, &(queries[ind].kl), sizeof(ui), hipMemcpyHostToDevice));
        chkerr(hipMemcpy(deviceGraph.lowerBoundSize + ind, &(queries[ind].N1), sizeof(ui), hipMemcpyHostToDevice));
        chkerr(hipMemcpy(deviceGraph.upperBoundSize + ind, & (queries[ind].N2), sizeof(ui), hipMemcpyHostToDevice));
        chkerr(hipMemcpy(deviceGraph.limitDoms + ind, &(queries[ind].limitDoms), sizeof(ui), hipMemcpyHostToDevice));

        chkerr(hipMemset(initialTask.globalCounter, 0, sizeof(ui)));
        chkerr(hipMemset(initialTask.entries, 0, INTOTAL_WARPS * sizeof(ui)));
        chkerr(hipMemset(deviceGraph.newOffset + ((n + 1) * ind),0,(n+1)*sizeof(ui)));


        if (queries[ind].kl <= 1)
          queries[ind].ubD = queries[ind].N2 - 1;
        else {
          for (ui d = 1; d <= queries[ind].N2; d++) {
            if (d == 1 || d == 2) {
              if (queries[ind].kl + d > queries[ind].N2) {
                queries[ind].ubD = d - 1;
                break;
              }
            } else {
              ui min_n = queries[ind].kl + d + 1 + floor(d / 3) * (queries[ind].kl - 2);
              if (queries[ind].N2 < min_n) {
                queries[ind].ubD = d - 1;
                break;
              }
            }
          }
        }
        maxN2 = mav(maxN2,queries[ind].N2);

        queries[ind].receiveTimer.restart();

        initialReductionRules << < BLK_NUM2, BLK_DIM2, sharedMemorySizeinitial >>> (deviceGenGraph, deviceGraph, initialTask, n, queries[ind].ubD, initialPartitionSize, ind);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("Intial Reduction ");

        ui globalCounter;
        chkerr(hipMemcpy( &globalCounter, initialTask.globalCounter, sizeof(ui), hipMemcpyDeviceToHost));

        ui writeWarp, ntasks, space;
        chkerr(hipMemcpy( &writeWarp, deviceTask.sortedIndex, sizeof(ui), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy( &ntasks, deviceTask.numTasks + writeWarp, sizeof(ui), hipMemcpyDeviceToHost));
        ui offsetPsize = partitionSize/factor;
        chkerr(hipMemcpy( &space, deviceTask.taskOffset + (writeWarp*offsetPsize + ntasks) , sizeof(ui), hipMemcpyDeviceToHost));
        if(globalCounter>=(partitionSize-space)){
          cout << "Intial Task > partition Size " << message << endl;
          continue;

        }


        CompressTask << < BLK_NUM2, BLK_DIM2 >>> (deviceGenGraph, deviceGraph, initialTask, deviceTask, initialPartitionSize, queries[ind].QID, ind, n,partitionSize,TOTAL_WARPS,factor);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("Compress ");

        thrust::inclusive_scan(thrust::device_ptr < ui > (deviceGraph.newOffset + ((n + 1) * ind)), thrust::device_ptr < ui > (deviceGraph.newOffset + ((n + 1) * (ind + 1))), thrust::device_ptr < ui > (deviceGraph.newOffset + ((n + 1) * ind)));
        hipDeviceSynchronize();

        numQueriesProcessing++;

        NeighborUpdate << < BLK_NUMS, BLK_DIM , sharedMemoryUpdateNeigh >>> (deviceGenGraph, deviceGraph,deviceTask, TOTAL_WARPS, ind, n, m,partitionSize,factor);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("Neighbor  ");
        thrust::device_ptr<ui> d_sortedIndex_ptr(deviceTask.sortedIndex);
        thrust::device_ptr<ui> d_mapping_ptr(deviceTask.mapping);

        thrust::device_vector<ui> d_temp_input(d_sortedIndex_ptr, d_sortedIndex_ptr + TOTAL_WARPS);

        thrust::sequence(thrust::device, d_sortedIndex_ptr, d_sortedIndex_ptr + TOTAL_WARPS);

        thrust::sort_by_key(thrust::device,
                            d_temp_input.begin(), d_temp_input.end(),
                            d_sortedIndex_ptr,
                            thrust::less<ui>());

        thrust::scatter(thrust::device,
                        thrust::make_counting_iterator<ui>(0),
                        thrust::make_counting_iterator<ui>(TOTAL_WARPS),
                        d_sortedIndex_ptr,
                        d_mapping_ptr);


        thrust::transform(thrust::device, d_mapping_ptr, d_mapping_ptr + TOTAL_WARPS, d_mapping_ptr, subtract_from(TOTAL_WARPS-1));
      messageQueueMutex.lock();
    }
    messageQueueMutex.unlock();

    if (numQueriesProcessing != 0) {
      chkerr(hipMemset(deviceGraph.flag,0, limitQueries * sizeof(ui)));
      chkerr(hipMemset(deviceTask.doms, 0, TOTAL_WARPS * partitionSize * sizeof(ui)));
      
      sharedMemorySizeTask = 2 * WARPS_EACH_BLK * sizeof(ui) + WARPS_EACH_BLK * sizeof(int) + WARPS_EACH_BLK * sizeof(double) + 2 * WARPS_EACH_BLK * sizeof(ui) + maxN2 * WARPS_EACH_BLK * sizeof(ui);

      ProcessTask << < BLK_NUMS, BLK_DIM, sharedMemorySizeTask >>> (
        deviceGenGraph, deviceGraph, deviceTask, partitionSize, factor,maxN2, n, m,dMAX,limitQueries);
      hipDeviceSynchronize();
      CUDA_CHECK_ERROR("Process Task");

      // This kernel identifies vertices dominated by ustar and sorts them in decreasing order of their connection score.
      FindDoms << < BLK_NUMS, BLK_DIM, sharedMemorySizeDoms >>> (
        deviceGenGraph, deviceGraph, deviceTask, partitionSize,factor, n, m,dMAX,limitQueries);
      hipDeviceSynchronize();
     CUDA_CHECK_ERROR("Find Doms");



      // This kernel writes new tasks, based on ustar and the dominating set, into the task array or buffer. It reads from the buffer and writes to the task array.
      //ui flag = 1 ;
      //chkerr(hipMemcpy( &flag, &deviceBuffer.outOfMemoryFlag, sizeof(ui),hipMemcpyDeviceToHost));
      Expand << < BLK_NUMS, BLK_DIM, sharedMemorySizeExpand >>> (
        deviceGenGraph, deviceGraph, deviceTask, deviceBuffer, partitionSize,factor, copyLimit, bufferSize, numTaskHost-numReadHost, readLimit, n, m, dMAX,limitQueries);
      hipDeviceSynchronize();
      CUDA_CHECK_ERROR("Expand ");
      RemoveCompletedTasks<<<BLK_NUMS, BLK_DIM>>>( deviceGraph,deviceTask, partitionSize,factor);
      hipDeviceSynchronize();
      CUDA_CHECK_ERROR("Remove Completed ");


      chkerr(hipMemcpy(&(outMemFlag), deviceBuffer.outOfMemoryFlag, sizeof(ui),hipMemcpyDeviceToHost));
     
      
      chkerr(hipMemcpy( &tempHost, deviceBuffer.temp, sizeof(ui),
        hipMemcpyDeviceToHost));
      chkerr(hipMemcpy( &numReadHost, deviceBuffer.numReadTasks, sizeof(ui),
        hipMemcpyDeviceToHost));
      chkerr(hipMemcpy( &numTaskHost, deviceBuffer.numTask, sizeof(ui),
        hipMemcpyDeviceToHost));


      chkerr(hipMemcpy(queryStopFlag, deviceGraph.flag, limitQueries * sizeof(ui), hipMemcpyDeviceToHost));

      for (ui i = 0; i < limitQueries; i++) {
          if ((queryStopFlag[i]==0) && (queries[i].solFlag==0)) {
            chkerr(hipMemcpy( & (queries[i].numRead), deviceGraph.numRead + i, sizeof(ui), hipMemcpyDeviceToHost));
            chkerr(hipMemcpy( & (queries[i].numWrite), deviceGraph.numWrite + i, sizeof(ui), hipMemcpyDeviceToHost));
            if ((queries[i].numRead == queries[i].numWrite)) {
              chkerr(hipMemcpy( & (queries[i].kl), deviceGraph.lowerBoundDegree + i, sizeof(ui), hipMemcpyDeviceToHost));
              
              stringstream ss;
              ss <<queries[i].N1<< "|" << queries[i].N2 << "|"<< queries[i].QID << "|"<< integer_to_string(queries[i].receiveTimer.elapsed()).c_str() << "|"<< queries[i].kl << "|"<<"0"<< "|"<<"0";
              writeOrAppend(fileName,ss.str());
              cout << "Found Solution : " << queries[i] << endl;
              queries[i].solFlag = 1;
              numQueriesProcessing--;
            }
          }


      }
       if(outMemFlag){
        for (ui i = 0; i < limitQueries; i++) {
          if ( queries[i].solFlag==0) {
          chkerr(hipMemcpy( & (queries[i].kl), deviceGraph.lowerBoundDegree + i, sizeof(ui), hipMemcpyDeviceToHost));
          
          stringstream ss;
          ss <<queries[i].N1<< "|" << queries[i].N2 << "|"<< queries[i].QID << "|"<< integer_to_string(queries[i].receiveTimer.elapsed()).c_str() << "|"<< queries[i].kl << "|"<<"2"<< "|"<<"0";
          writeOrAppend(fileName,ss.str());
          cout <<"Buffer out of memory !"<<endl;
          cout << "Found Solution : " << queries[i] << endl;
          queries[i].solFlag = 1;
          numQueriesProcessing--;
            
          }
       }
       break;
        
      }
      

      if (numTaskHost == numReadHost) {
        chkerr(hipMemset(deviceBuffer.numTask, 0, sizeof(ui)));
        chkerr(hipMemset(deviceBuffer.numReadTasks, 0, sizeof(ui)));
        chkerr(hipMemset(deviceBuffer.temp, 0, sizeof(ui)));
        chkerr(hipMemset(deviceBuffer.writeMutex, 0, sizeof(ui)));
        chkerr(hipMemset(deviceBuffer.readMutex, 0, sizeof(ui)));
        chkerr(hipMemset(deviceBuffer.taskOffset, 0, (numReadHost + 1) * sizeof(ui)));
      }

      // If the number of tasks written to the buffer exceeds the number read at this level, left shift the tasks that were written but not read to the start of the array.
      if ((numReadHost < numTaskHost) && (numReadHost > 0)) {
        chkerr(hipMemcpy( & startOffset, deviceBuffer.taskOffset + numReadHost,
          sizeof(ui), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy( & endOffset, deviceBuffer.taskOffset + numTaskHost, sizeof(ui),
          hipMemcpyDeviceToHost));

        thrust::transform(
          thrust::device_ptr < ui > (deviceBuffer.taskOffset + numReadHost),
          thrust::device_ptr < ui > (deviceBuffer.taskOffset + numTaskHost + 1),
          thrust::device_ptr < ui > (deviceBuffer.taskOffset),
          subtract_functor(startOffset));

        chkerr(hipMemset(deviceBuffer.taskOffset + (numTaskHost - numReadHost + 1), 0,
          numReadHost * sizeof(ui)));

        thrust::copy(thrust::device_ptr < ui > (deviceBuffer.size + numReadHost),
          thrust::device_ptr < ui > (deviceBuffer.size + numTaskHost),
          thrust::device_ptr < ui > (deviceBuffer.size));

        thrust::copy(thrust::device_ptr < ui > (deviceBuffer.queryIndicator + numReadHost),
          thrust::device_ptr < ui > (deviceBuffer.queryIndicator + numTaskHost),
          thrust::device_ptr < ui > (deviceBuffer.queryIndicator));

        thrust::copy(
          thrust::device_ptr < ui > (deviceBuffer.taskList + startOffset),
          thrust::device_ptr < ui > (deviceBuffer.taskList + endOffset),
          thrust::device_ptr < ui > (deviceBuffer.taskList));

        thrust::copy(
          thrust::device_ptr < ui > (deviceBuffer.statusList + startOffset),
          thrust::device_ptr < ui > (deviceBuffer.statusList + endOffset),
          thrust::device_ptr < ui > (deviceBuffer.statusList));

        int justCheck = (int)(numTaskHost - numReadHost);

        chkerr(hipMemcpy(deviceBuffer.numTask, & justCheck, sizeof(ui),
          hipMemcpyHostToDevice));
        chkerr(hipMemcpy(deviceBuffer.temp, & justCheck, sizeof(ui),
          hipMemcpyHostToDevice));
        chkerr(hipMemset(deviceBuffer.writeMutex, 0, sizeof(ui)));
        chkerr(hipMemset(deviceBuffer.numReadTasks, 0, sizeof(ui)));

        chkerr(hipMemset(deviceBuffer.readMutex, 0, sizeof(ui)));
      }
      c++;
      thrust::device_ptr<ui> d_sortedIndex_ptr(deviceTask.sortedIndex);
      thrust::device_ptr<ui> d_mapping_ptr(deviceTask.mapping);

      thrust::device_vector<ui> d_temp_input(d_sortedIndex_ptr, d_sortedIndex_ptr + TOTAL_WARPS);

      thrust::sequence(thrust::device, d_sortedIndex_ptr, d_sortedIndex_ptr + TOTAL_WARPS);

      thrust::sort_by_key(thrust::device,
                          d_temp_input.begin(), d_temp_input.end(),
                          d_sortedIndex_ptr,
                          thrust::less<ui>());

      thrust::scatter(thrust::device,
                      thrust::make_counting_iterator<ui>(0),
                      thrust::make_counting_iterator<ui>(TOTAL_WARPS),
                      d_sortedIndex_ptr,
                      d_mapping_ptr);


      thrust::transform(thrust::device, d_mapping_ptr, d_mapping_ptr + TOTAL_WARPS, d_mapping_ptr, subtract_from(TOTAL_WARPS-1));

    }

    if(c==200){
        for (ui i = 0; i < limitQueries; i++) {
          if ( queries[i].solFlag==0) {
          chkerr(hipMemcpy( & (queries[i].kl), deviceGraph.lowerBoundDegree + i, sizeof(ui), hipMemcpyDeviceToHost));
          
          stringstream ss;
          ss <<queries[i].N1<< "|" << queries[i].N2 << "|"<< queries[i].QID << "|"<< integer_to_string(queries[i].receiveTimer.elapsed()).c_str() << "|"<< queries[i].kl << "|"<<"1"<< "|"<<"0";
          writeOrAppend(fileName,ss.str());
          cout <<"Levels > 200 !"<<endl;
          cout << "Found Solution : " << queries[i] << endl;
          queries[i].solFlag = 1;
          numQueriesProcessing--;
            
          }
       }
       break;
        
    }

    if(serverQuit && (numQueriesProcessing==0))
    break;

  }
}

int main(int argc, const char * argv[]) {
  if (argc != 8) {
    cout << "Server wrong input parameters!" << endl;
    exit(1);
  }

  const char * filepath = argv[1]; // Path to the graph file. The graph should be represented as an edge list with tab (\t) separators
  partitionSize = atoi(argv[2]); // Defines the partition size, in number of elements, that a single warp will read from and write to.
  bufferSize = atoi(argv[3]); // Specifies the size, in number of elements, where warps will write in case the partition overflows.
  copyLimit = stod(argv[4]); // Specifies that only warps with at most this percentage of their partition space filled will read from the buffer and write to their partition.
  readLimit = atoi(argv[5]); // Maximum number of tasks a warp with an empty partition can read from the buffer.
  limitQueries = atoi(argv[6]);
  factor = atoi(argv[7]); 
  
  graphPath = argv[1];
  size_t pos = graphPath.find_last_of("/\\");
  fileName = (pos != string::npos) ? graphPath.substr(pos + 1) : graphPath;

  fileName = "./results/maxdeg/" + fileName;

  if (!fileExists(fileName)) {
      string header = "N1|N2|QID|Time|Degree|Overtime|Heu";
      ofstream file;
      file.open(fileName, ios::app);
          if (file.is_open()) {
              file << header << endl;
              file.close();
          }
  }

  queries = new queryData[limitQueries];
  for(ui i =0; i < limitQueries;i ++ ){
     queryData query;
    queries[i] = query;
  }

  load_graph(filepath);




  core_decomposition_linear_list();

  memoryAllocationGenGraph(deviceGenGraph);
  memeoryAllocationGraph(deviceGraph, limitQueries);

  totalQuerry = 0;
  q_dist = new ui[n];
  outMemFlag = 0;

  maxN2 = 0;
  if (n <= WARPSIZE) {
        BLK_DIM2 = WARPSIZE;
        BLK_NUM2 = 1;
  } else if (n <= BLK_NUMS) {
      BLK_DIM2 = std::ceil(static_cast<float>(n) / WARPSIZE) * WARPSIZE;
      BLK_NUM2 = 1;
  } else {
      BLK_DIM2 = BLK_DIM;
      BLK_NUM2 = std::min(BLK_NUMS, static_cast<int>(std::ceil(static_cast<float>(n) / BLK_DIM2)));
  }

  INTOTAL_WARPS = (BLK_NUM2 * BLK_DIM2) / WARPSIZE;

  initialPartitionSize = static_cast<ui>(std::ceil(static_cast<float>(n) / INTOTAL_WARPS));
  memoryAllocationinitialTask(initialTask, INTOTAL_WARPS, initialPartitionSize);
  memoryAllocationTask(deviceTask, TOTAL_WARPS, partitionSize, limitQueries,factor);
  memoryAllocationBuffer(deviceBuffer, bufferSize,limitQueries,factor);


  sharedMemorySizeinitial = INTOTAL_WARPS * sizeof(ui);
  sharedMemoryUpdateNeigh = WARPS_EACH_BLK * sizeof(ui);

  queryStopFlag = new ui[limitQueries];

  memset(queryStopFlag,0, limitQueries * sizeof(ui));

  sharedMemorySizeDoms = WARPS_EACH_BLK * sizeof(ui);
  sharedMemorySizeExpand = WARPS_EACH_BLK * sizeof(ui);

  numTaskHost = 0;
  numReadHost = 0;
  tempHost = 0;
  startOffset = 0;
  endOffset = 0;

  numQueriesProcessing = 0;
  thread listener(listenForMessages);
	thread processor(processMessages);
  listener.join();
  processor.join();
  hipDeviceSynchronize();
  cout<<"End"<<endl;
  freeGenGraph(deviceGenGraph);
  freeGraph(deviceGraph);
  freeInterPointer(initialTask);
  freeTaskPointer(deviceTask);
  freeBufferPointer(deviceBuffer);
  hipDeviceSynchronize();
  hipDeviceReset();


  return 0;
}
