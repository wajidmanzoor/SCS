#include "../inc/Intialize.h"


inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cout<<hipGetErrorString(code)<<std::endl;
        exit(-1);
    }
}


void memoryAllocationGenGraph(deviceGraphGenPointers &G){
    chkerr(hipMalloc((void**)&(G.core), n * sizeof(ui)));
    chkerr(hipMemcpy(G.core, core, n * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.degree), n * sizeof(ui)));
    chkerr(hipMemcpy(G.degree, degree, n * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.offset), (n+1) * sizeof(ui)));
    chkerr(hipMemcpy(G.offset, pstart, (n+1) * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.neighbors), (2*m) * sizeof(ui)));
    chkerr(hipMemcpy(G.neighbors, edges, (2*m) * sizeof(ui), hipMemcpyHostToDevice));

}

void memeoryAllocationGraph(deviceGraphPointers &G, ui totalQueries){

    chkerr(hipMalloc((void**)&(G.degree),totalQueries* n * sizeof(ui)));
    chkerr(hipMalloc((void**)&(G.distance),totalQueries* n * sizeof(ui)));

    chkerr(hipMalloc((void**)&(G.lowerBoundDegree), totalQueries*sizeof(ui)));
    chkerr(hipMalloc((void**)&(G.lowerBoundSize), totalQueries*sizeof(ui)));
    chkerr(hipMalloc((void**)&(G.upperBoundSize), totalQueries*sizeof(ui)));
    chkerr(hipMalloc((void**)&(G.limitDoms), totalQueries*sizeof(ui)));
    chkerr(hipMalloc((void**)&(G.flag), totalQueries*sizeof(ui)));
    chkerr(hipMalloc((void**)&(G.numRead), totalQueries*sizeof(ui)));
    chkerr(hipMalloc((void**)&(G.numWrite), totalQueries*sizeof(ui)));

    chkerr(hipMalloc((void **)&(G.newNeighbors), totalQueries*(2 * m) * sizeof(ui)));
    chkerr(hipMalloc((void **)&(G.newOffset), totalQueries*(n + 1) * sizeof(ui)));
    chkerr(hipMemset(G.newOffset,0, totalQueries*(n + 1) * sizeof(ui)));
}

void memoryAllocationinitialTask(deviceInterPointers &p, ui numWraps,ui psize){
    chkerr(hipMalloc((void**)&(p.initialTaskList), numWraps*psize*sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.globalCounter), sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.entries),numWraps* sizeof(ui)));
}

void memoryAllocationTask(deviceTaskPointers &p, ui numWraps, ui pSize, ui totalQueries){
    chkerr(hipMalloc((void**)&(p.taskList), numWraps*pSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&p.statusList, numWraps*pSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.degreeInR), numWraps*pSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.degreeInC), numWraps*pSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.taskOffset), numWraps*pSize*sizeof(ui)));
    chkerr(hipMemset(p.taskOffset,0, numWraps*pSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.ustar), numWraps*pSize*sizeof(int)));

    chkerr(hipMalloc((void**)&(p.size), numWraps*pSize*sizeof(ui)));
    chkerr(hipMemset(p.size,0,numWraps*pSize*sizeof(ui)));

     chkerr(hipMalloc((void**)&(p.doms), numWraps*pSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.cons), numWraps*pSize*sizeof(double)));

    chkerr(hipMalloc((void**)&(p.queryIndicator), numWraps*pSize*sizeof(ui)));



}

void memoryAllocationBuffer(deviceBufferPointers &p,ui bufferSize){

    chkerr(hipMalloc((void**)&(p.taskOffset), bufferSize*sizeof(ui)));
    chkerr(hipMemset(p.taskOffset,0, bufferSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.taskList), bufferSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&p.statusList, bufferSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.degreeInR), bufferSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.degreeInC), bufferSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.size), bufferSize*sizeof(ui)));
    chkerr(hipMemset(p.size,0,bufferSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.numTask), sizeof(ui)));
    chkerr(hipMemset(p.numTask,0,sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.temp), sizeof(ui)));
    chkerr(hipMemset(p.temp,0,sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.numReadTasks), sizeof(ui)));
    chkerr(hipMemset(p.numReadTasks,0,sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.writeMutex), sizeof(ui)));
    chkerr(hipMemset(p.writeMutex,0,sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.readMutex), sizeof(ui)));
    chkerr(hipMemset(p.readMutex,0,sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.queryIndicator), bufferSize*sizeof(ui)));
    
    chkerr(hipMalloc((void**)&(p.outOfMemoryFlag), sizeof(ui)));
    chkerr(hipMemset(p.outOfMemoryFlag,0,sizeof(ui)));

}

void freeGenGraph(deviceGraphGenPointers &p){
    chkerr(hipFree(p.offset));
    chkerr(hipFree(p.neighbors));
    chkerr(hipFree(p.degree));
    chkerr(hipFree(p.core));


}

void freeGraph(deviceGraphPointers &p){

    chkerr(hipFree(p.degree));
    chkerr(hipFree(p.distance));
    chkerr(hipFree(p.newNeighbors));
    chkerr(hipFree(p.newOffset));
    chkerr(hipFree(p.lowerBoundDegree));
    chkerr(hipFree(p.lowerBoundSize));
    chkerr(hipFree(p.upperBoundSize));
    chkerr(hipFree(p.limitDoms));
    chkerr(hipFree(p.flag));
    chkerr(hipFree(p.numRead));
    chkerr(hipFree(p.numWrite));


}


void freeInterPointer(deviceInterPointers &p){
    chkerr(hipFree(p.initialTaskList));
    chkerr(hipFree(p.globalCounter));
    chkerr(hipFree(p.entries));

}

void freeTaskPointer(deviceTaskPointers &p){
    chkerr(hipFree(p.taskList));
    chkerr(hipFree(p.statusList));
    chkerr(hipFree(p.taskOffset));
    chkerr(hipFree(p.size));
    chkerr(hipFree(p.degreeInR));
    chkerr(hipFree(p.degreeInC));
    chkerr(hipFree(p.ustar));
    chkerr(hipFree(p.doms));
    chkerr(hipFree(p.cons));
    chkerr(hipFree(p.queryIndicator));


}

void freeBufferPointer(deviceBufferPointers &p){
    chkerr(hipFree(p.taskOffset));
    chkerr(hipFree(p.taskList));
    chkerr(hipFree(p.statusList));
    chkerr(hipFree(p.degreeInC));
    chkerr(hipFree(p.degreeInR));
    chkerr(hipFree(p.size));
    chkerr(hipFree(p.numTask));
    chkerr(hipFree(p.temp));
    chkerr(hipFree(p.numReadTasks));
    chkerr(hipFree(p.writeMutex));
    chkerr(hipFree(p.readMutex));
    chkerr(hipFree(p.queryIndicator));

}

