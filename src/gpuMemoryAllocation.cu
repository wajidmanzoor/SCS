#include "../inc/ListLinearHeap.h"

inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cout<<hipGetErrorString(code)<<std::endl;
        exit(-1);
    }
}


void memoryAllocationGraph(deviceGraphPointers &G){
    chkerr(hipMalloc((void**)&(G.core), n * sizeof(ui)));
    chkerr(hipMemcpy(G.core, core, n * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.degree), n * sizeof(ui)));
    chkerr(hipMemcpy(G.degree, degree, n * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.offset), (n+1) * sizeof(ui)));
    chkerr(hipMemcpy(G.offset, pstart, (n+1) * sizeof(ui), hipMemcpyHostToDevice));


    chkerr(hipMalloc((void**)&(G.neighbors), (2*m) * sizeof(ui)));
    chkerr(hipMemcpy(G.neighbors, edges, (2*m) * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.distance), n * sizeof(ui)));
    chkerr(hipMemcpy(G.distance, q_dist, n * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.lowerBoundDegree), sizeof(ui)));
    chkerr(hipMemcpy(G.lowerBoundDegree, &kl,sizeof(ui),hipMemcpyHostToDevice));
}

void memoryAllocationinitialTask(deviceInterPointers &p, ui numWraps,ui psize){
    chkerr(hipMalloc((void**)&(p.initialTaskList), numWraps*psize*sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.globalCounter), sizeof(ui)));
    chkerr(hipMemset(p.globalCounter,0,sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.entries),numWraps* sizeof(ui)));
}

void memoryAllocationTask(deviceTaskPointers &p, ui numWraps, ui pSize){
    chkerr(hipMalloc((void**)&(p.taskList), numWraps*pSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&p.statusList, numWraps*pSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.degreeInR), numWraps*pSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.degreeInC), numWraps*pSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.taskOffset), numWraps*pSize*sizeof(ui)));
    chkerr(hipMemset(p.taskOffset,0, numWraps*pSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.ustar), numWraps*pSize*sizeof(int)));

    chkerr(hipMalloc((void**)&(p.size), numWraps*pSize*sizeof(ui)));
    chkerr(hipMemset(p.size,0,numWraps*pSize*sizeof(ui)));

     chkerr(hipMalloc((void**)&(p.doms), numWraps*pSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.cons), numWraps*pSize*sizeof(double)));


    chkerr(hipMalloc((void**)&(p.flag),sizeof(bool)));


}

void memoryAllocationBuffer(deviceBufferPointers &p,ui numWraps, ui pSize){

    chkerr(hipMalloc((void**)&(p.taskOffset), numWraps*pSize*sizeof(ui)));
    chkerr(hipMemset(p.taskOffset,0, numWraps*pSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.taskList), numWraps*pSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&p.statusList, numWraps*pSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.degreeInR), numWraps*pSize*sizeof(ui)));
    chkerr(hipMalloc((void**)&(p.degreeInC), numWraps*pSize*sizeof(ui)));



    chkerr(hipMalloc((void**)&(p.size), numWraps*pSize*sizeof(ui)));
    chkerr(hipMemset(p.size,0,numWraps*pSize*sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.numTask), sizeof(ui)));
    chkerr(hipMemset(p.numTask,0,sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.temp), sizeof(ui)));
    chkerr(hipMemset(p.temp,0,sizeof(ui)));

    chkerr(hipMalloc((void**)&(p.numReadTasks), sizeof(ui)));
    chkerr(hipMemset(p.numReadTasks,0,sizeof(ui)));


}

void freeGraph(deviceGraphPointers &p){
    chkerr(hipFree(p.core));
    chkerr(hipFree(p.degree));
    chkerr(hipFree(p.distance));
    chkerr(hipFree(p.lowerBoundDegree));
    chkerr(hipFree(p.neighbors));
    chkerr(hipFree(p.offset));

}


void freeInterPointer(deviceInterPointers &p){
    chkerr(hipFree(p.entries));
    chkerr(hipFree(p.globalCounter));
    chkerr(hipFree(p.initialTaskList));

}

void freeTaskPointer(deviceTaskPointers &p){
    chkerr(hipFree(p.size));
    chkerr(hipFree(p.statusList));
    chkerr(hipFree(p.taskList));
    chkerr(hipFree(p.taskOffset));
    chkerr(hipFree(p.ustar));
    //chkerr(hipFree(p.flag));


}

