#include "../inc/ListLinearHeap.h"


inline void chkerr(hipError_t code) {
  if (code != hipSuccess) {
    std::cout << hipGetErrorString(code) << std::endl;
    exit(-1);
  }
}

void memoryAllocationGraph(deviceGraphPointers &G) {
  chkerr(hipMalloc((void **)&(G.core), n * sizeof(ui)));
  chkerr(hipMemcpy(G.core, core, n * sizeof(ui), hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(G.degree), n * sizeof(ui)));
  chkerr(hipMemcpy(G.degree, degree, n * sizeof(ui), hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(G.offset), (n + 1) * sizeof(ui)));
  chkerr(hipMemcpy(G.offset, pstart, (n + 1) * sizeof(ui),
                    hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(G.neighbors), (2 * m) * sizeof(ui)));
  chkerr(hipMemcpy(G.neighbors, edges, (2 * m) * sizeof(ui),
                    hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(G.distance), n * sizeof(ui)));
  chkerr(
      hipMemcpy(G.distance, q_dist, n * sizeof(ui), hipMemcpyHostToDevice));

  chkerr(hipMalloc((void **)&(G.lowerBoundDegree), sizeof(ui)));
  chkerr(
      hipMemcpy(G.lowerBoundDegree, &kl, sizeof(ui), hipMemcpyHostToDevice));
}

void memoryAllocationinitialTask(deviceInterPointers &p, ui numWraps,
                                 ui psize) {
  chkerr(
      hipMalloc((void **)&(p.initialTaskList), numWraps * psize * sizeof(ui)));
  chkerr(hipMalloc((void **)&(p.globalCounter), sizeof(ui)));
  chkerr(hipMemset(p.globalCounter, 0, sizeof(ui)));
  chkerr(hipMalloc((void **)&(p.entries), numWraps * sizeof(ui)));
}

void memoryAllocationTask(deviceTaskPointers &p, ui numWraps, ui pSize) {
  chkerr(hipMalloc((void **)&(p.taskList), numWraps * pSize * sizeof(ui)));
  chkerr(hipMalloc((void **)&p.statusList, numWraps * pSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.degreeInR), numWraps * pSize * sizeof(ui)));
  chkerr(hipMalloc((void **)&(p.degreeInC), numWraps * pSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.taskOffset), numWraps * pSize * sizeof(ui)));
  chkerr(hipMemset(p.taskOffset, 0, numWraps * pSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.ustar), numWraps * pSize * sizeof(int)));

  chkerr(hipMalloc((void **)&(p.size), numWraps * pSize * sizeof(ui)));
  chkerr(hipMemset(p.size, 0, numWraps * pSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.doms), numWraps * pSize * sizeof(ui)));
  chkerr(hipMalloc((void **)&(p.cons), numWraps * pSize * sizeof(double)));

  chkerr(hipMalloc((void **)&(p.flag), sizeof(bool)));
}

void memoryAllocationBuffer(deviceBufferPointers &p, ui bufferSize) {
  chkerr(hipMalloc((void **)&(p.taskOffset), bufferSize * sizeof(ui)));
  chkerr(hipMemset(p.taskOffset, 0, bufferSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.taskList), bufferSize * sizeof(ui)));
  chkerr(hipMalloc((void **)&p.statusList, bufferSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.degreeInR), bufferSize * sizeof(ui)));
  chkerr(hipMalloc((void **)&(p.degreeInC), bufferSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.size), bufferSize * sizeof(ui)));
  chkerr(hipMemset(p.size, 0, bufferSize * sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.numTask), sizeof(ui)));
  chkerr(hipMemset(p.numTask, 0, sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.temp), sizeof(ui)));
  chkerr(hipMemset(p.temp, 0, sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.numReadTasks), sizeof(ui)));
  chkerr(hipMemset(p.numReadTasks, 0, sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.writeMutex), sizeof(ui)));
  chkerr(hipMemset(p.writeMutex, 0, sizeof(ui)));

  chkerr(hipMalloc((void **)&(p.readMutex), sizeof(ui)));
  chkerr(hipMemset(p.readMutex, 0, sizeof(ui)));
}

void freeGraph(deviceGraphPointers &p) {
  chkerr(hipFree(p.offset));
  chkerr(hipFree(p.neighbors));
  chkerr(hipFree(p.degree));
  chkerr(hipFree(p.distance));
  chkerr(hipFree(p.core));
  chkerr(hipFree(p.lowerBoundDegree));
}

void freeInterPointer(deviceInterPointers &p) {
  chkerr(hipFree(p.initialTaskList));
  chkerr(hipFree(p.globalCounter));
  chkerr(hipFree(p.entries));
}

void freeTaskPointer(deviceTaskPointers &p) {
  chkerr(hipFree(p.taskList));
  chkerr(hipFree(p.statusList));
  chkerr(hipFree(p.taskOffset));
  chkerr(hipFree(p.size));
  chkerr(hipFree(p.degreeInR));
  chkerr(hipFree(p.degreeInC));
  chkerr(hipFree(p.ustar));
  chkerr(hipFree(p.doms));
  chkerr(hipFree(p.cons));
  chkerr(hipFree(p.flag));
}

void freeBufferPointer(deviceBufferPointers &p) {
  chkerr(hipFree(p.taskOffset));
  chkerr(hipFree(p.taskList));
  chkerr(hipFree(p.statusList));
  chkerr(hipFree(p.degreeInC));
  chkerr(hipFree(p.degreeInR));
  chkerr(hipFree(p.size));
  chkerr(hipFree(p.numTask));
  chkerr(hipFree(p.temp));
  chkerr(hipFree(p.numReadTasks));
  chkerr(hipFree(p.writeMutex));
  chkerr(hipFree(p.readMutex));
}

